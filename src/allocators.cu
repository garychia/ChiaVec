#include "allocators.hpp"

#include <hip/hip_runtime.h>

namespace ChiaVec {
namespace Memory {
void *DefaultAllocator::operator()(std::size_t size) {
  void *ptr = nullptr;

  if (0 == size) {
    return ptr;
  }

  hipHostMalloc(&ptr, size);
  return ptr;
}

void DefaultAllocator::release(void *ptr) {
  if (!ptr) {
    return;
  }
  hipHostFree(ptr);
}

void *DefaultAllocator::resize(void *ptr, std::size_t oldSize,
                               std::size_t newSize) {
  if (newSize > oldSize) {
    void *newPtr = nullptr;
    hipHostMalloc(&newPtr, newSize);
    hipMemcpy(newPtr, ptr, oldSize, hipMemcpyHostToHost);
    hipHostFree(ptr);
    ptr = newPtr;
  }
  return ptr;
}

void DefaultAllocator::copy(void *dst, const void *src, std::size_t size,
                            bool dstOnHost, bool srcOnHost) {
  if (dstOnHost) {
    if (srcOnHost) {
      hipMemcpy(dst, src, size, hipMemcpyHostToHost);
    } else {
      hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
    }
  } else {
    if (srcOnHost) {
      hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
    } else {
      hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice);
    }
  }
}

void *DefaultCudaAllocator::operator()(std::size_t size) {
  void *ptr = nullptr;
  if (0 == size) {
    return ptr;
  }
  hipMalloc(&ptr, size);
  return ptr;
}

void DefaultCudaAllocator::release(void *ptr) {
  if (ptr) {
    hipFree(ptr);
  }
}

void *DefaultCudaAllocator::resize(void *ptr, std::size_t oldSize,
                                   std::size_t newSize) {
  if (newSize > oldSize) {
    void *newPtr = nullptr;
    hipHostMalloc(&newPtr, newSize);
    hipMemcpy(newPtr, ptr, oldSize, hipMemcpyHostToHost);
    hipHostFree(ptr);
    ptr = newPtr;
  }
  return ptr;
}

void DefaultCudaAllocator::copy(void *dst, const void *src, std::size_t size,
                                bool dstOnHost, bool srcOnHost) {
  if (dstOnHost) {
    if (srcOnHost) {
      hipMemcpy(dst, src, size, hipMemcpyHostToHost);
    } else {
      hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
    }
  } else {
    if (srcOnHost) {
      hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
    } else {
      hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice);
    }
  }
}

} // namespace Memory
} // namespace ChiaVec